#include "hip/hip_runtime.h"
#include "cuda_rand_generation.h"

#include <hiprand.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <locale>
#include <stdio.h>
#include <string>
#include <thrust/copy.h>
#include <thrust/distance.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scan.h>
#include <thrust/transform_reduce.h>
#include <time.h>

typedef unsigned long long ulonglong;

using namespace std;

// Comma grouping for number output.
struct GroupedNumbers : public std::numpunct<char> {
 protected:
  string do_grouping() const { return "\003"; }
};

template <typename T>
struct shift_functor
{
    explicit shift_functor(T shift) : shift_(shift) {}

    T shift_;

    __device__
    T operator()(const T& x) const { 
      return x + shift_;
    }
};

CUDARandaomWalkGenerator::CUDARandaomWalkGenerator(
  const ulonglong seed, const float lastValue, const ulonglong offset,
  const bool generateFiles) : kGenerateFiles_(generateFiles) {
  // Get properties of default device.
  hipDeviceProp_t properties;
  hipError_t status = hipGetDeviceProperties(&properties, 0);
  if (status != hipSuccess) {
    // TODO(Bilson): Do something later about unsupported devices.
  }
  
  if (kGenerateFiles_)
    hostData = new float[kBatchSizeInterval];

  cout.imbue(locale(cout.getloc(), new GroupedNumbers));
  // Allocation of device memory with an extra location for random walk
  // initialization.
  status = hipMalloc((void **)&devData, kBatchSizeInterval * sizeof(float));
  if (status != hipSuccess) {
    std::cout << "Error in device allocation.\n";
    // TODO(Bilson): Need to do something about this failure.
  }

  lastValue_ = lastValue;
  nextOffset_ = offset;

  // Next file prefix number.
  nextFileNumber_ = 0;

  // CURAND library random number generator (RNG).
   // Create generator.
  if (hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT) !=
      HIPRAND_STATUS_SUCCESS) {
    std::cout << "Error in creating generator.\n";
    // TODO(Bilson): Need to do something about this failure.
  }
  // Set seed.
  if (hiprandSetPseudoRandomGeneratorSeed(gen_, seed) !=
      HIPRAND_STATUS_SUCCESS) {
    std::cout << "Error in setting seed.\n";
    // TODO(Bilson): Need to do something about this failure.
  }
  // Set offset.
  if (hiprandSetGeneratorOffset(gen_, nextOffset_) !=
      HIPRAND_STATUS_SUCCESS) {
    std::cout << "Error in setting seed.\n";
  }
}

CUDARandaomWalkGenerator::~CUDARandaomWalkGenerator() {
  if (kGenerateFiles_)
    delete[] hostData;

  if (hipFree(devData) != hipSuccess) {
    std::cout << "Error in device allocation.\n";
    // TODO(Bilson): Need to do something about this failure.
  }

  if (hiprandDestroyGenerator(gen_) != HIPRAND_STATUS_SUCCESS) {
    std::cout << "Error in creating numbers.\n";
    // TODO(Bilson): Need to do something about this failure.
  }
}

void CUDARandaomWalkGenerator::GenerateFilename(const char *filePrefix,
                                                const unsigned int index,
                                                char *fileName) {
  sprintf(fileName, "%s%05u", filePrefix, index);
}

ulonglong CUDARandaomWalkGenerator::GenerateRandomWalkArray(
    const ulonglong points, float *buffer) {
  // Thrust pointer.
  thrust::device_ptr<float> thrustPtr(devData);
  ulonglong pointsGenerated = 0;
  while (pointsGenerated < points) {
    ulonglong currentBatchCount =
      pointsGenerated + kBatchSizeInterval > points ?
        points - pointsGenerated : kBatchSizeInterval;

    const bool kIsOdd = currentBatchCount & 0x00000001;
    if (kIsOdd) {
      // Must be a multiple of two.
      currentBatchCount++;
    }

    // Create random numbers.
    hiprandStatus_t status =
      hiprandGenerateNormal(gen_, devData, currentBatchCount, 0, 1);
    if (status != HIPRAND_STATUS_SUCCESS) {
      std::cout << "Error in creating numbers.\n";
      return false;
    }
    
    // Dont need the extra random value.
    if (kIsOdd)
      currentBatchCount--;

    // Prefix sum for random walk.
    thrust::inclusive_scan(thrustPtr, thrustPtr + currentBatchCount,
                           thrustPtr);

    // Shift by initial value.
    thrust::transform(thrustPtr, thrustPtr + currentBatchCount, thrustPtr,
      shift_functor<float>(lastValue_));

    // Copy new segment back to host.
    if (hipMemcpy(buffer + pointsGenerated, devData,
        currentBatchCount * sizeof(float), hipMemcpyDeviceToHost)
        != hipSuccess) {
      std::cout << "Error in copying data to host.\n";
      return false;
    }

    // Save last value for next segment.
    lastValue_ = buffer[pointsGenerated + currentBatchCount - 1];

    pointsGenerated += currentBatchCount;
  }
  nextOffset_ += pointsGenerated + 1;
  return pointsGenerated;
}

bool CUDARandaomWalkGenerator::GenerateRandomWalkFile(
    const char* filePrefix, const ulonglong fileCount,
    const ulonglong pointsPerFile) {
  char *fileName = new char[512];

  clock_t start = clock();
  cout << "Creating " << fileCount << " files.\n";
  for (int i = 0; i < fileCount; ++i) {
    GenerateFilename(filePrefix, nextFileNumber_++, fileName);
    cout << "Beginning file " << fileName << endl;
    // Create file and update next offset value.
    nextOffset_ += PopulateFile(fileName, pointsPerFile) + 1;
  }

  clock_t end = clock();

  // Report times and clean up.
  cout << "Elapsed generation minutes: "
       << (end - start) / (double) CLOCKS_PER_SEC / 60.0
       << endl;
  delete[] fileName;

  cout << "Last point: " << lastValue_ << endl;
  cout << "Next offset: " << nextOffset_ << endl;
  return true;
}

ulonglong CUDARandaomWalkGenerator::PopulateFile(const char* fileName,
                                                 const ulonglong points) {
  // Thrust pointer.
  thrust::device_ptr<float> thrustPtr(devData);
  // Output stream.
  ofstream outfile(fileName, ofstream::out | ofstream::binary);
  ulonglong pointsGenerated = 0;
  cout << "Creating " << points << " points.\n";
  while (pointsGenerated < points) {
    ulonglong currentBatchCount =
      pointsGenerated + kBatchSizeInterval > points ?
        points - pointsGenerated :
        kBatchSizeInterval;
    
    const bool kIsOdd = currentBatchCount & 0x00000001;
    if (kIsOdd) {
      // Must be a multiple of two.
      currentBatchCount++;
    }

    // Create random numbers.
    if (hiprandGenerateNormal(gen_, devData, currentBatchCount, 0, 1) !=
          HIPRAND_STATUS_SUCCESS) {
      std::cout << "Error in creating numbers.\n";
      return false;
    }

    // Dont need the extra random value.
    if (kIsOdd)
      currentBatchCount--;

    // Prefix sum for random walk.
    thrust::inclusive_scan(thrustPtr, thrustPtr + currentBatchCount,
                           thrustPtr);

    // Shift by initial value.
    thrust::transform(thrustPtr, thrustPtr + currentBatchCount, thrustPtr,
      shift_functor<float>(lastValue_));

    // Copy new segment back to host.
    if (hipMemcpy(hostData, devData, currentBatchCount * sizeof(float),
                   hipMemcpyDeviceToHost)
          != hipSuccess) {
      std::cout << "Error in copying data to host.\n";
      return false;
    }

    // Save last value for next segment.
    lastValue_ = hostData[currentBatchCount - 1];

    outfile.write(reinterpret_cast<char *>(hostData),
                  currentBatchCount * sizeof(float));
    pointsGenerated += currentBatchCount;
    cout << "\rPoints generated: " << pointsGenerated;
  }
  cout << endl;
  outfile.close();
  return pointsGenerated;
}
